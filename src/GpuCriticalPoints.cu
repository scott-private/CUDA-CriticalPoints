/*
============================================================================
Author      : Scott Fu
Date        : 17/06/2020
Copyright   : scottfu@foxmail.com
File Name   : CriticalPoints.cpp
============================================================================
*/
#include <cassert>
#include <iostream>
#include <vector>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "CriticalPoints.h"
#include "KernelCriticalPoints.h"


GpuCPFinder::GpuCPFinder(const char* filename) : BaseCPFinder(filename)
{
    initialization();
}

GpuCPFinder::~GpuCPFinder()
{
    checkCudaErrors(hipFree(d_tets));
    checkCudaErrors(hipFree(d_outtets));
    checkCudaErrors(hipFree(d_outpoints));
    checkCudaErrors(hipFree(d_outtets2));
    checkCudaErrors(hipFree(d_outpoints2));
    checkCudaErrors(hipFree(d_tetsize));
    checkCudaErrors(hipFree(d_pointsize));
    checkCudaErrors(hipFree(d_queryvec));
}


void GpuCPFinder::initialization()
{
    // Exactinit
    const int PredParaNum = 14;	        // the size of hostConst
    REAL hostConst[14];					// constants used in exact CUDA CCW test

	//exact computation
	REAL *d_constants;
    checkCudaErrors(hipMalloc((void **)&d_constants, PredParaNum * sizeof(REAL)) ); 
    cudaExactinit<<<1,1>>>(d_constants); 
	checkCudaErrors(hipMemcpy(hostConst, d_constants, PredParaNum * sizeof(REAL), hipMemcpyDeviceToHost) ); 
	
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(deviceConst), hostConst, PredParaNum * sizeof(REAL)) ); 
    checkCudaErrors(hipFree(d_constants));

    // Tet init
    tetnum = seqtets.size() / 4;

    checkCudaErrors(hipMalloc((void **)&d_tets, seqtets.size() * sizeof(PointVelocity3D)) ); 
    checkCudaErrors(hipMemcpy(d_tets, seqtets.data(), seqtets.size() * sizeof(PointVelocity3D), hipMemcpyHostToDevice) ); 

    checkCudaErrors(hipMalloc((void **)&d_outtets, tetnum * sizeof(int)) ); 
    checkCudaErrors(hipMalloc((void **)&d_outpoints, tetnum * 4 * sizeof(PointTet)) ); 
    checkCudaErrors(hipMalloc((void **)&d_outtets2, tetnum * sizeof(int)) ); 
    checkCudaErrors(hipMalloc((void **)&d_outpoints2, tetnum * 4 * sizeof(PointTet)) ); 

    checkCudaErrors(hipMalloc((void **)&d_tetsize, sizeof(int)) ); 
    checkCudaErrors(hipMalloc((void **)&d_pointsize, sizeof(int)) );

    checkCudaErrors(hipMemset(d_tetsize,0,sizeof(int))) ; 
    checkCudaErrors(hipMemset(d_pointsize,0,sizeof(int))) ;
    
    cudaInit<<<100,256>>>(d_outtets, d_outpoints, tetnum);

    Vector3D query = {0.0,0.0,0.0};
    checkCudaErrors(hipMalloc((void **)&d_queryvec, sizeof(Vector3D)) );
    checkCudaErrors(hipMemcpy(d_queryvec, &query, sizeof(Vector3D), hipMemcpyHostToDevice) ); 
}


void GpuCPFinder::search()
{
    cudaSearch<<<100,256>>>(d_tets, tetnum, d_queryvec, d_outtets, d_outpoints);
    checkCudaErrors(hipDeviceSynchronize());

    cudaCompress<<<100,256>>>(d_outtets, d_outpoints, tetnum, d_outtets2, d_outpoints2, d_tetsize, d_pointsize);

    int pointSize = 0, tetSize = 0;
    checkCudaErrors(hipMemcpy(&tetSize, d_tetsize, sizeof(int), hipMemcpyDeviceToHost) ); 
    checkCudaErrors(hipMemcpy(&pointSize, d_pointsize, sizeof(int), hipMemcpyDeviceToHost) ); 
    // std::cout << pointSize << " " << tetSize << std::endl;
    criticalTets.resize(tetSize);
    criticalPoints.resize(pointSize);
    checkCudaErrors(hipMemcpy(criticalTets.data(), d_outtets2, tetSize * sizeof(int), hipMemcpyDeviceToHost) ); 
    checkCudaErrors(hipMemcpy(criticalPoints.data(), d_outpoints2, pointSize * sizeof(PointTet), hipMemcpyDeviceToHost) ); 
}